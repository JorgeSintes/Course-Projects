extern "C"
{


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

#define BLOCK_SIZE 16

__global__ 
void
matmultgpu3_colwise(int m, int n, int k, double *A, double *B, double *C) {
    // Bad one
 double Cvalue1 = 0.0, 
        Cvalue2 = 0.0;

  int col = 2*(blockIdx.x*blockDim.x+threadIdx.x);
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  
  int e;

  if ((row < m) && (col < (n - 1))) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[row*k+e]*B[e*n+col+1];
    }
        
    C[row*n+col]=Cvalue1;
    C[row*n+col+1]=Cvalue2;
  }

  else if ((row < m) && (col == (n - 1))) {
    for(e=0;e<k;++e)
        Cvalue1+=A[row*k+e]*B[e*n+col];
        
        C[row*n+col]=Cvalue1;
  }
	
}

__global__ 
void
matmultgpu3_rowwise(int m, int n, int k, double *A, double *B, double *C) {
    // This is the good one!!!
    	
 double Cvalue1 = 0.0, 
        Cvalue2 = 0.0;

  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int row = 2*(blockIdx.y*blockDim.y+threadIdx.y);
  
  int e;

  if ((row < m-1) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
  }

  else if ((row == m -1) && (col < n)) {
    for(e=0;e<k;++e)
        Cvalue1+=A[row*k+e]*B[e*n+col];
        
        C[row*n+col]=Cvalue1;
  }
	
}



void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C){

  double *d_A, *d_B, *d_C;

  int blocky;
  int sizeA = m * k *sizeof(double);
  int sizeB = k * n *sizeof(double);
  int sizeC = m * n *sizeof(double);

  double time1, time2, elapsed;


  // Declare the number of threads
  dim3 numOfThreadsPerBlock;
  numOfThreadsPerBlock.x = BLOCK_SIZE;
  numOfThreadsPerBlock.y = BLOCK_SIZE;

  // Initializing for colwise
  // blocky = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  // dim3 numOfBlocks;
  // numOfBlocks.x = (blocky+1)/2;
  // numOfBlocks.y = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);

  // Initializing for rowwise
  blocky = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);
  dim3 numOfBlocks;
  numOfBlocks.x = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  numOfBlocks.y = (blocky+1)/2;  
  
  // Allocate memory on the device
  hipMalloc((void**)&d_A, sizeA);
  hipMalloc((void**)&d_B, sizeB);
  hipMalloc((void**)&d_C, sizeC);

  time1 = omp_get_wtime();

  // Copy the values over
  hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
  
  time2 = omp_get_wtime();

  matmultgpu3_rowwise<<<numOfBlocks, numOfThreadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
  hipDeviceSynchronize();

  elapsed = omp_get_wtime() - time2;
  printf("Kernel time: %f\n", elapsed);

  hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

  elapsed = omp_get_wtime() - time1;
  printf("Kernel+copy time: %f\n", elapsed);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}
