extern "C"
{


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

#define BLOCK_SIZE 16

__global__ 
void
matmultgpu4_rowwise(int m, int n, int k, double *A, double *B, double *C) {
    // This is the good one!!!
    	
 double Cvalue1 = 0.0, 
        Cvalue2 = 0.0,
        Cvalue3 = 0.0,
        Cvalue4 = 0.0,
        Cvalue5 = 0.0,
        Cvalue6 = 0.0,
        Cvalue7 = 0.0;

  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int row = 7*(blockIdx.y*blockDim.y+threadIdx.y);
  
  int e;

  if ((row < m-6) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
        Cvalue3 += A[(row+2)*k+e]*B[e*n+col];
        Cvalue4 += A[(row+3)*k+e]*B[e*n+col];
        Cvalue5 += A[(row+4)*k+e]*B[e*n+col];
        Cvalue6 += A[(row+5)*k+e]*B[e*n+col];
        Cvalue7 += A[(row+6)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
    C[(row+2)*n+col]=Cvalue3;
    C[(row+3)*n+col]=Cvalue4;
    C[(row+4)*n+col]=Cvalue5;
    C[(row+5)*n+col]=Cvalue6;
    C[(row+6)*n+col]=Cvalue7;
  }

  else if ((row == m-6) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
        Cvalue3 += A[(row+2)*k+e]*B[e*n+col];
        Cvalue4 += A[(row+3)*k+e]*B[e*n+col];
        Cvalue5 += A[(row+4)*k+e]*B[e*n+col];
        Cvalue6 += A[(row+5)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
    C[(row+2)*n+col]=Cvalue3;
    C[(row+3)*n+col]=Cvalue4;
    C[(row+4)*n+col]=Cvalue5;
    C[(row+5)*n+col]=Cvalue6;
  }

  else if ((row == m-5) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
        Cvalue3 += A[(row+2)*k+e]*B[e*n+col];
        Cvalue4 += A[(row+3)*k+e]*B[e*n+col];
        Cvalue5 += A[(row+4)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
    C[(row+2)*n+col]=Cvalue3;
    C[(row+3)*n+col]=Cvalue4;
    C[(row+4)*n+col]=Cvalue5;
  }
  
  else if ((row == m-4) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
        Cvalue3 += A[(row+2)*k+e]*B[e*n+col];
        Cvalue4 += A[(row+3)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
    C[(row+2)*n+col]=Cvalue3;
    C[(row+3)*n+col]=Cvalue4;
  }

  else if ((row == m-3) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
        Cvalue3 += A[(row+2)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
    C[(row+2)*n+col]=Cvalue3;
  }

  else if ((row == m-2) && (col < n)) {
    for(e=0;e<k;++e) {
        Cvalue1 += A[row*k+e]*B[e*n+col];
        Cvalue2 += A[(row+1)*k+e]*B[e*n+col];
    }
        
    C[row*n+col]=Cvalue1;
    C[(row+1)*n+col]=Cvalue2;
  }

  else if ((row == m -1) && (col < n)) {
    for(e=0;e<k;++e)
        Cvalue1+=A[row*k+e]*B[e*n+col];
        
        C[row*n+col]=Cvalue1;
  }
	
}



void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C){

  double *d_A, *d_B, *d_C;

  int blocky;
  int sizeA = m * k *sizeof(double);
  int sizeB = k * n *sizeof(double);
  int sizeC = m * n *sizeof(double);

  double time1, time2, elapsed;

  // Declare the number of threads
  dim3 numOfThreadsPerBlock;
  numOfThreadsPerBlock.x = BLOCK_SIZE;
  numOfThreadsPerBlock.y = BLOCK_SIZE;

  // Initializing for colwise
  // blocky = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  // dim3 numOfBlocks;
  //   numOfBlocks.x = (blocky+6)/7;
  // numOfBlocks.y = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);

  // Initializing for rowwise
  blocky = (m+numOfThreadsPerBlock.y-1)/(numOfThreadsPerBlock.y);
  dim3 numOfBlocks;
  numOfBlocks.x = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  numOfBlocks.y = (blocky+6)/7;
  
  // Allocate memory on the device
  hipMalloc((void**)&d_A, sizeA);
  hipMalloc((void**)&d_B, sizeB);
  hipMalloc((void**)&d_C, sizeC);

  time1 = omp_get_wtime();

  // Copy the values over
  hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

  time2 = omp_get_wtime();

  matmultgpu4_rowwise<<<numOfBlocks, numOfThreadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
  hipDeviceSynchronize();

  elapsed = omp_get_wtime() - time2;
  printf("Kernel time: %f\n", elapsed);

  hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

  elapsed = omp_get_wtime() - time1;
  printf("Kernel+copy time: %f\n", elapsed);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}
