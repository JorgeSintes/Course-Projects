extern "C"{


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

__global__ 
void
matmultgpu1(int m, int n, int k, double *A, double *B, double *C) {
    	
     int i1,i2,i3;

    
    for(i1 = 0; i1< m; i1++){
    	for(i2 = 0; i2 < n; i2++){    
             C[i1*n+i2]=0;        
	    for(i3 = 0; i3 < k; i3++){ 
		C[i1*n+i2]+=A[i1*k+i3]*B[i3*n+i2];
	     }
        }
    }
	
}



void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C){

  double   *d_A, *d_B, *d_C;

  int sizeA = m * k *sizeof(double);
  int sizeB = k * n *sizeof(double);
  int sizeC = m * n *sizeof(double);

  double time1, time2, elapsed;
 

  //Alloc memory on the device
  hipMalloc((void**)&d_A,sizeA);
  hipMalloc((void**)&d_B,sizeB);
  hipMalloc((void**)&d_C,sizeC);

  time1 = omp_get_wtime();

  hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,sizeB,hipMemcpyHostToDevice);
   
  time2 = omp_get_wtime();
  
  matmultgpu1<<<1,1>>>(m,n,k,d_A,d_B,d_C);
  hipDeviceSynchronize();

  elapsed = omp_get_wtime() - time2;
  printf("Kernel time: %f\n", elapsed);

  hipMemcpy(C,d_C,sizeC,hipMemcpyDeviceToHost);

  elapsed = omp_get_wtime() - time1;
  printf("Kernel+copy time: %f\n", elapsed);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);



}
}

