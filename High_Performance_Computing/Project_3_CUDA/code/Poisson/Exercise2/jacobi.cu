/* jacobi.c - Poisson problem in 3d
 * 
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

__device__ void print_matrix2(double*** A, int N){
	int i,j,k;
	for (i=0; i<N; i++){
		printf("\n %d -th Layer \n", i);
		for(j=0; j<N; j++){
			for(k=0; k<N; k++){	
				printf("%lf \t", A[i][j][k]);
			}
		printf("\n");
		}
	}
}

__global__ void jacobi_gpu2(double*** u, double***prev_u, double*** f, int N, double step_width, double denominator) {

	double temp;
	
	int j_index=threadIdx.y + blockIdx.y*blockDim.y;
    int k_index= threadIdx.x + blockIdx.x*blockDim.x;
	int i_index=threadIdx.z + blockIdx.z*blockDim.z; 

	
	//printf("%d %d %d \n", j_index, k_index, i_index);
	
	if ((j_index<N-2) && (k_index<N-2) && (i_index<N-2)){
			temp=prev_u[i_index][j_index+1][k_index+1] + prev_u[i_index+2][j_index+1][k_index+1]+ 
				prev_u[i_index+1][j_index][k_index+1] + prev_u[i_index+1][j_index+2][k_index+1] + 
				prev_u[i_index+1][j_index+1][k_index]+ prev_u[i_index+1][j_index+1][k_index+2] + step_width*step_width*f[i_index+1][j_index+1][k_index+1];
			u[i_index+1][j_index+1][k_index+1]=temp*denominator;
		}
	
	//printf("On the GPU we now have matrix:\n");
	//print_matrix2(u,N);
}
