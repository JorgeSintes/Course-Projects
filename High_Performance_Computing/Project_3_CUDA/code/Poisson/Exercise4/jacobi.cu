/* jacobi.c - Poisson problem in 3d
 * 
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

__inline__ __device__
double warpReduceSum(double value) {
	for (int i = 16; i > 0; i /= 2)
	value += __shfl_down_sync(-1, value, i);
	return value;
} 

__inline__ __device__
double blockReduceSum(double value) {
	__shared__ double smem[32]; // Max 32 warp sums
	if ((threadIdx.x < warpSize)&&(threadIdx.y < warpSize)&&(threadIdx.z < warpSize))
		smem[threadIdx.x] = 0;
	__syncthreads();
	value = warpReduceSum(value);
	if (threadIdx.x % warpSize == 0)
		smem[threadIdx.x / warpSize] = value;
	__syncthreads();
	if (threadIdx.x < warpSize)
		value = smem[threadIdx.x];
 return warpReduceSum(value);
} 

__device__ void print_matrix2(double*** A, int N){
	int i,j,k;
	for (i=0; i<N; i++){
		printf("\n %d -th Layer \n", i);
		for(j=0; j<N; j++){
			for(k=0; k<N; k++){	
				printf("%lf \t", A[i][j][k]);
			}
		printf("\n");
		}
	}
}

__global__ void jacobi_gpu4(double*** u, double***prev_u, double*** f, int N, double step_width, double denominator, double* norm) {
	double temp;
	int j_index=threadIdx.y + blockIdx.y*blockDim.y;
    int k_index= threadIdx.x + blockIdx.x*blockDim.x;
	int i_index=threadIdx.z + blockIdx.z*blockDim.z; 
	double temp2;
	double value=0;
	
	//printf("%d %d %d \n", j_index, k_index, i_index);
	
	if ((j_index<N-2) && (k_index<N-2) && (i_index<N-2)){
			temp=prev_u[i_index][j_index+1][k_index+1] + prev_u[i_index+2][j_index+1][k_index+1]+ 
				prev_u[i_index+1][j_index][k_index+1] + prev_u[i_index+1][j_index+2][k_index+1] + 
				prev_u[i_index+1][j_index+1][k_index]+ prev_u[i_index+1][j_index+1][k_index+2] + step_width*step_width*f[i_index+1][j_index+1][k_index+1];
			u[i_index+1][j_index+1][k_index+1]=temp*denominator;
			temp2 =  (prev_u[i_index+1][j_index+1][k_index+1] - temp*denominator)*(prev_u[i_index+1][j_index+1][k_index+1] - temp*denominator);
			value=blockReduceSum(temp2);
			//value = warpReduceSum(temp2);
			if ((threadIdx.x % warpSize == 0)&&(threadIdx.y % warpSize == 0)&&(threadIdx.z % warpSize == 0)) atomicAdd(norm, value); 		
			//atomicAdd(norm,temp2);
		}
	
	//printf("On the GPU we now have matrix:\n");
	//print_matrix2(u,N);
}
